#include "hip/hip_runtime.h"
/* GPU-based finite difference on 3-D grid
*/
/*
  Copyright (C) 2014  Xi'an Jiaotong University (Pengliang Yang)

  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA

  Reference:
    Micikevicius, Paulius. "3D finite difference computation on GPUs
    using CUDA." Proceedings of 2nd Workshop on General Purpose 
    Processing on Graphics Processing Units. ACM, 2009.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

extern "C" {
#include <rsf.h>
}

#ifndef PI
#define PI 	SF_PI
#endif
#define BlockSize1 16// tile size in 1st-axis
#define BlockSize2 16// tile size in 2nd-axis
#define radius 	4// half of the order in space

void sf_check_gpu_error (const char *msg) 
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) { 
	sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err)); 
	exit(0);   
    }
}

__constant__ float stencil[radius+1]={-205.0/72.0,8.0/5.0,-1.0/5.0,8.0/315.0,-1.0/560.0};

__global__ void cuda_ricker_wavelet(float *wlt, float fm, float dt, int nt)
/*< generate ricker wavelet with time deley >*/
{
	int it=threadIdx.x+blockDim.x*blockIdx.x;
    	if (it<nt){
	  float tmp = PI*fm*fabsf(it*dt-1.0/fm);//delay the wavelet to exhibit all waveform
	  tmp *=tmp;
	  wlt[it]= (1.0-2.0*tmp)*expf(-tmp);// ricker wavelet at time: t=nt*dt
	}
}



__global__ void cuda_set_sg(int *szxy, int szbeg, int sxbeg, int sybeg, int jsz, int jsx, int jsy, int ns, int n1, int n2, int n3)
/*< set the positions of sources and geophones in whole domain >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
	int nn1=n1+2*radius;
	int nn2=n2+2*radius;
    	if (id<ns) szxy[id]=(szbeg+id*jsz+radius)+nn1*(sxbeg+id*jsx+radius)+nn1*nn2*(sybeg+id*jsy+radius);
}


__global__ void cuda_add_source(bool add, float *p, float *source, int *szxy, int ns)
/*< add/subtract sources: length of source[]=ns, index stored in szxy[] >*/
{
  int id=threadIdx.x+blockIdx.x*blockDim.x;

  if(id<ns){
    if(add){
      p[szxy[id]]+=source[id];
    }else{
      p[szxy[id]]-=source[id];
    }
  }
}

__global__ void cuda_step_fd3d(float *p0, float *p1, float *vv, float _dz2, float _dx2, float _dy2, int n1, int n2, int n3)
/*< step forward: 3-D FD, order=8 >*/
{
    bool validr = true;
    bool validw = true;
    const int gtid1 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gtid2 = blockIdx.y * blockDim.y + threadIdx.y;
    const int ltid1 = threadIdx.x;
    const int ltid2 = threadIdx.y;
    const int work1 = blockDim.x;
    const int work2 = blockDim.y;
    __shared__ float tile[BlockSize2 + 2 * radius][BlockSize1 + 2 * radius];

    const int stride2 = n1 + 2 * radius;
    const int stride3 = stride2 * (n2 + 2 * radius);

    int inIndex = 0;
    int outIndex = 0;

    // Advance inputIndex to start of inner volume
    inIndex += radius * stride2 + radius;

    // Advance inputIndex to target element
    inIndex += gtid2 * stride2 + gtid1;

    float infront[radius];
    float behind[radius];
    float current;

    const int t1 = ltid1 + radius;
    const int t2 = ltid2 + radius;

    // Check in bounds
    if ((gtid1 >= n1 + radius) ||(gtid2 >= n2 + radius)) validr = false;
    if ((gtid1 >= n1) || (gtid2 >= n2)) validw = false;

    // Preload the "infront" and "behind" data
    for (int i = radius - 2 ; i >= 0 ; i--)
    {
        if (validr) behind[i] = p1[inIndex];
        inIndex += stride3;
    }

    if (validr)	current = p1[inIndex];

    outIndex = inIndex;
    inIndex += stride3;

    for (int i = 0 ; i < radius ; i++)
    {
	if (validr) infront[i] = p1[inIndex];
        inIndex += stride3;
    }

    // Step through the zx-planes
#pragma unroll 9
    for (int i3 = 0 ; i3 < n3 ; i3++)
    {
        // Advance the slice (move the thread-front)
        for (int i = radius - 1 ; i > 0 ; i--) behind[i] = behind[i - 1];

        behind[0] = current;
        current = infront[0];
#pragma unroll 4
        for (int i = 0 ; i < radius - 1 ; i++) infront[i] = infront[i + 1];

        if (validr) infront[radius - 1] = p1[inIndex];

        inIndex += stride3;
        outIndex += stride3;
        __syncthreads();

        // Update the data slice in the local tile
        // Halo above & below
        if (ltid2 < radius)
        {
            tile[ltid2][t1]                  = p1[outIndex - radius * stride2];
            tile[ltid2 + work2 + radius][t1] = p1[outIndex + work2 * stride2];
        }

        // Halo left & right
        if (ltid1 < radius)
        {
            tile[t2][ltid1]                  = p1[outIndex - radius];
            tile[t2][ltid1 + work1 + radius] = p1[outIndex + work1];
        }

        tile[t2][t1] = current;
        __syncthreads();

        // Compute the output value
	float c1, c2, c3;
        c1=c2=c3=stencil[0]*current;        
#pragma unroll 4
        for (int i=1; i <= radius ; i++)
        {
	  c1 +=stencil[i]*(tile[t2][t1-i]+ tile[t2][t1+i]);
	  c2 +=stencil[i]*(tile[t2-i][t1]+ tile[t2+i][t1]);
	  c3 +=stencil[i]*(infront[i-1]  + behind[i-1]  ); 
        }
	c1*=_dz2;	
	c2*=_dx2;
	c3*=_dy2;
        if (validw) p0[outIndex]=2.0*p1[outIndex]-p0[outIndex]+vv[outIndex]*(c1+c2+c3);
    }
}

void velocity_transform(float *v0, float*vv, float dt, int n1, int n2, int n3)
 /*< velocit2 transform: vv=v0*dt; vv<--vv^2 >*/
{
  int i1, i2, i3, nn1, nn2, nn3;
  float tmp;

  nn1=n1+2*radius;
  nn2=n2+2*radius;
  nn3=n3+2*radius;

  for(i3=0; i3<n3; i3++){
    for(i2=0; i2<n2; i2++){
      for(i1=0; i1<n1; i1++){
	tmp=v0[i1+n1*i2+n1*n2*i3]*dt;
	vv[(i1+radius)+nn1*(i2+radius)+nn1*nn2*(i3+radius)]=tmp*tmp;
      }
    }
  }  

    for         (i3=0; i3<nn3; 	i3++) {
	for     (i2=0; i2<nn2; 	i2++) {
	    for (i1=0; i1<radius;i1++) {
		vv[i1+nn1*i2+nn1*nn2*i3]=vv[radius+nn1*i2+nn1*nn2*i3];
		vv[(nn1-i1-1)+nn1*i2+nn1*nn2]=vv[(nn1-radius-1)+nn1*i2+nn1*nn2];
	    }
	}
    }


    for         (i3=0; i3<nn3; 	i3++) {
	for     (i2=0; i2<radius;i2++) {
	    for (i1=0; i1<nn1; 	i1++) {
		vv[i1+nn1*i2+nn1*nn2*i3]=vv[i1+nn1*radius+nn1*nn2*i3];
		vv[i1+nn1*(nn2-i2-1)+nn1*nn2*i3]=vv[i1+nn1*(nn2-radius-1)+nn1*nn2*i3];
	    }
	}
    }

    for         (i3=0; i3<radius;i3++) {
	for     (i2=0; i2<nn2; 	i2++) {
	    for (i1=0; i1<nn1; 	i1++) {
		vv[i1+nn1*i2+nn1*nn2*i3]=vv[i1+nn1*i2+nn1*nn2*radius];
		vv[i1+nn1*i2+nn1*nn2*(nn3-1-i3)]=vv[i1+nn1*i2+nn1*nn2*(nn3-radius-1)];
	    }
	}
    }
}


void window3d(float *a, float *b, int n1, int n2, int n3)
/*< window a 3d subvolume >*/
{
	int i1, i2, i3, nn1, nn2;
	nn1=n1+2*radius;
	nn2=n2+2*radius;
	
	for(i3=0; i3<n3; i3++)
	for(i2=0; i2<n2; i2++)
	for(i1=0; i1<n1; i1++)
	{
		a[i1+n1*i2+n1*n2*i3]=b[(i1+radius)+nn1*(i2+radius)+nn1*nn2*(i3+radius)];
	}
}


int main(int argc, char* argv[])
{
	bool verb;
	int nz, nx, ny, nnz, nnx, nny, ns, nt, kt, it, is, szbeg, sxbeg, sybeg, jsz, jsx, jsy;
	int *d_szxy;
	float dz, dx, dy, fm, dt, _dz2, _dx2, _dy2;
	float *v0, *vv, *d_wlt, *d_vv, *d_p0, *d_p1, *ptr;
	sf_file Fv, Fw;

    	sf_init(argc,argv);
	Fv=sf_input("in");
	Fw=sf_output("out");

    	if (!sf_getbool("verb",&verb)) verb=false; /* verbosit2 */
    	if (!sf_histint(Fv,"n1",&nz)) sf_error("No n1= in input");
    	if (!sf_histint(Fv,"n2",&nx)) sf_error("No n2= in input");
    	if (!sf_histint(Fv,"n3",&ny)) sf_error("No n3= in input");
    	if (!sf_histfloat(Fv,"d1",&dz)) sf_error("No d1= in input");
    	if (!sf_histfloat(Fv,"d2",&dx)) sf_error("No d2= in input");
    	if (!sf_histfloat(Fv,"d3",&dy)) sf_error("No d3= in input");
   	if (!sf_getint("nt",&nt))  sf_error("nt required");
	/* total number of time steps */
    	if (!sf_getint("kt",&kt)) sf_error("kt required");
	/* record wavefield at time kt */
   	if (!sf_getfloat("dt",&dt))  sf_error("dt required");
	/* time sampling interval */
   	if (!sf_getfloat("fm",&fm))  fm=20;
	/* dominant frequency of Ricker wavelet */
   	if (!sf_getint("ns",&ns))  ns=1;
	/* number of sources */
	if (!sf_getint("szbeg",&szbeg)) sf_error("No szbeg");
	/* source beginning of z-axis */
	if (!sf_getint("sxbeg",&sxbeg)) sf_error("No sxbeg");
	/* source beginning of x-axis */
	if (!sf_getint("sybeg",&sybeg)) sf_error("No sybeg");
	/* source beginning of y-axis */
	if (!sf_getint("jsz",&jsz)) sf_error("No jsz");
	/* source jump interval in z-axis */
	if (!sf_getint("jsx",&jsx)) sf_error("No jsx");
	/* source jump interval in x-axis */
	if (!sf_getint("jsy",&jsy)) sf_error("No jsy");
	/* source jump interval in y-axis */

	sf_putint(Fw,"n1",nz);
	sf_putint(Fw,"n2",nx);
	sf_putint(Fw,"n3",ny);

	_dz2=1.0/(dz*dz);
	_dx2=1.0/(dx*dx);
	_dy2=1.0/(dy*dy);
	nnz=nz+2*radius;
	nnx=nx+2*radius;
	nny=ny+2*radius;
    	v0=(float*)malloc(nz*nx*ny*sizeof(float));
    	vv=(float*)malloc(nnz*nnx*nny*sizeof(float));
	sf_floatread(v0, nz*nx*ny, Fv);// read velocit2 model v0
	velocity_transform(v0, vv, dt, nz, nx, ny);// init

    	hipSetDevice(0);// initialize device, default device=0;
	sf_check_gpu_error("Failed to initialize device!");

	dim3 dimg, dimb;
	dimg.x=(nz+BlockSize1-1)/BlockSize1;
	dimg.y=(nx+BlockSize2-1)/BlockSize2;
	dimb.x=BlockSize1;
	dimb.y=BlockSize2;

	/* allocate memory on device */
	hipMalloc(&d_wlt, nt*sizeof(float));
	hipMalloc(&d_vv, nnz*nnx*nny*sizeof(float));
	hipMalloc(&d_p0, nnz*nnx*nny*sizeof(float));
	hipMalloc(&d_p1, nnz*nnx*nny*sizeof(float));
	hipMalloc(&d_szxy, ns*sizeof(int));
	sf_check_gpu_error("Failed to allocate memory for variables!");

	cuda_ricker_wavelet<<<(nt+511)/512, 512>>>(d_wlt, fm, dt, nt);
	hipMemcpy(d_vv, vv, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
	cuda_set_sg<<<1, ns>>>(d_szxy, szbeg, sxbeg, sybeg, jsz, jsx, jsy, ns, nz, nx, ny);

	float mstimer;
	clock_t t0, t1;
	hipEvent_t start, stop;
  	hipEventCreate(&start);	
	hipEventCreate(&stop);
	for(is=0; is<ns; is++){
	  hipEventRecord(start);

	  hipMemset(d_p0, 0, nnz*nnx*nny*sizeof(float));
	  hipMemset(d_p1, 0, nnz*nnx*nny*sizeof(float));
	  for(it=0; it<nt; it++){
	    cuda_add_source<<<1,1>>>(true, d_p1, &d_wlt[it], &d_szxy[is], 1);
	    cuda_step_fd3d<<<dimg,dimb>>>(d_p0, d_p1, d_vv, _dz2, _dx2, _dy2, nz, nx, ny);
	    ptr=d_p0; d_p0=d_p1; d_p1=ptr;//toggle buffers

	    if(it==kt){
	      t0 = clock();

	      hipMemcpy(vv, d_p0, nnz*nnx*nny*sizeof(float), hipMemcpyDeviceToHost);
	      window3d(v0, vv, nz, nx, ny);
	      sf_floatwrite(v0, nz*nx*ny, Fw);	  

 	      t1 = clock();
 	      sf_warning("save the volume: %f (s)", ((float)(t1-t0))/CLOCKS_PER_SEC); 	
	    }

	    sf_warning("it=%d",it);
	  }
	  hipEventRecord(stop);
          hipEventSynchronize(stop);
  	  hipEventElapsedTime(&mstimer, start, stop);
    	  sf_warning("%d shot finished: %g (s)",is+1, mstimer*1.e-3);
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);

	/* free memory on device */
	hipFree(d_wlt);
	hipFree(d_vv);
	hipFree(d_p0);
	hipFree(d_p1);
	hipFree(d_szxy);
	free(v0);
	free(vv);

    	exit (0);
}
